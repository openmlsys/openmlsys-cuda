#include "hip/hip_runtime.h"
#include "util.cuh"

namespace {
template <typename LayoutTile, typename LayoutBlock, typename LayoutThread>
__global__ void gemmKernel(const float *__restrict__ A,
                           const float *__restrict__ B, float *__restrict__ C,
                           float alpha, float beta, unsigned M, unsigned N,
                           unsigned K) {
  constexpr unsigned ratio = sizeof(openmlsys::float4) / sizeof(float);
  unsigned int m = threadIdx.x * LayoutThread::m + LayoutTile::m * blockIdx.x;
  unsigned int n = threadIdx.y * LayoutThread::n + LayoutTile::n * blockIdx.y;
  openmlsys::Tensor2D<const float> pA{A, M, K};
  pA.addOffset(m, 0);
  openmlsys::Tensor2D<const openmlsys::float4> pB{B, K, N / ratio};
  pB.addOffset(0, n / ratio);
  openmlsys::Tensor2D<openmlsys::float4> pC{C, M, N / ratio};
  pC.addOffset(m, n / ratio);
  if (!pC.validOffset(0, 0)) return;

  const unsigned iterationA = LayoutTile::m / LayoutBlock::m / LayoutThread::m;
  const unsigned iterationB = LayoutTile::n / LayoutBlock::n / LayoutThread::n;
  const unsigned intervalA = LayoutTile::m / iterationA;
  const unsigned intervalB = LayoutTile::n / iterationB;
  openmlsys::float4 c[iterationA][iterationB][4];
  memset(c, 0, sizeof(c));
  for (unsigned k = 0; k < K; ++k) {
#pragma unroll
    for (unsigned iterA = 0; iterA < iterationA; ++iterA) {
#pragma unroll
      for (unsigned iterB = 0; iterB < iterationB; ++iterB) {
        openmlsys::float4 fragmentA{};
#pragma unroll
        for (unsigned i = 0; i < ratio; ++i) {
          fragmentA[i] = pA(i + iterA * intervalA, k);
        }
        openmlsys::float4 fragmentB = pB(k, iterB * intervalB / ratio);

#pragma unroll
        for (unsigned i = 0; i < ratio; ++i) {
          c[iterA][iterB][i] = c[iterA][iterB][i] + fragmentB * fragmentA[i];
        }
      }
    }
  }

#pragma unroll
  for (auto &termA : c) {
#pragma unroll
    for (auto &termB : termA) {
#pragma unroll
      for (auto &term : termB) {
        term = term * alpha;
      }
    }
  }

#pragma unroll
  for (unsigned iterA = 0; iterA < iterationA; ++iterA) {
#pragma unroll
    for (unsigned iterB = 0; iterB < iterationB; ++iterB) {
#pragma unroll
      for (unsigned i = 0; i < ratio; ++i) {
        openmlsys::float4 result{c[iterA][iterB][i]};
        if (beta != 0) {
          result = result +
                   pC(i + iterA * intervalA, iterB * intervalB / ratio) * beta;
        }
        pC(i + iterA * intervalA, iterB * intervalB / ratio) = result;
      }
    }
  }
}
}  // namespace

void gemmUseTile(const float *deviceAPtr, const float *deviceBPtr,
                 float *deviceCPtr, float alpha, float beta, unsigned M,
                 unsigned N, unsigned K) {
  using LayoutTile = openmlsys::Layout<128, 128, 16>;
  using LayoutBlock = openmlsys::Layout<16, 16>;
  using LayoutThread = openmlsys::Layout<4, 4>;

  dim3 block(LayoutBlock::m, LayoutBlock::n);
  dim3 grid((M * LayoutTile::m / LayoutBlock::m - 1) / LayoutBlock::m + 1,
            (N * LayoutTile::n / LayoutBlock::n - 1) / LayoutBlock::n + 1);

  gemmKernel<LayoutTile, LayoutBlock, LayoutThread><<<grid, block>>>(
      deviceAPtr, deviceBPtr, deviceCPtr, alpha, beta, M, N, K);
}
