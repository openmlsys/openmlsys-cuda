#include "hip/hip_runtime.h"
#include <Eigen/Core>
#include <ctime>
#include <omp.h>

__global__ void gemmKernel(const float *A, const float *B, float *C,
                           float alpha, float beta, unsigned M, unsigned N,
                           unsigned K) {
  unsigned int m = threadIdx.x + blockDim.x * blockIdx.x;
  unsigned int n = threadIdx.y + blockDim.y * blockIdx.y;
  if (m >= M || n >= N)
    return;
  float c = 0;
  for (unsigned k = 0; k < K; ++k) {
    c += A[m * K + k] * B[k * N + n];
  }
  c = c * alpha;
  float result = c;
  if (beta != 0) {
    result = result + C[m * N + n] * beta;
  }
  C[m * N + n] = result;
}

void gemmNaive(const float *A, const float *B, float *C, float alpha,
               float beta, unsigned M, unsigned N, unsigned K) {
  dim3 block(32, 32);
  dim3 grid((M - 1) / block.x + 1, (N - 1) / block.y + 1);

  gemmKernel<<<grid, block>>>(A, B, C, alpha, beta, M, N, K);
}

using namespace Eigen;

int main() {
  int gpu_rank = 0;
  hipDeviceProp_t deviceProp{};
  hipGetDeviceProperties(&deviceProp, gpu_rank);
  hipSetDevice(gpu_rank);
  printf("GPU %s status: ", deviceProp.name);
  double boostFrequency = deviceProp.clockRate / 1e6;
  int fp32CoresNum = 640;
  double peakPerformance = boostFrequency * fp32CoresNum * 2;
  printf("clock rate %.3f GHz, FP32 cores num %d, FP32 peak throughput %.3f "
         "GFLOPS\n",
         boostFrequency, fp32CoresNum, peakPerformance);
  omp_set_num_threads(omp_get_num_procs());
  unsigned M = 1024, N = 1024, K = 1024;
  float alpha = 1., beta = 0.;
  float *deviceAPrt, *deviceBPtr, *deviceCPtr;
  Matrix<float, Dynamic, Dynamic, RowMajor> A{M, K}, B{K, N}, C{M, N};
  A.setRandom();
  B.setRandom();
  C.setRandom();
  hipMalloc(&deviceAPrt, M * K * sizeof(float));
  hipMemcpy(deviceAPrt, A.data(), M * K * sizeof(float),
             hipMemcpyHostToDevice);
  hipMalloc(&deviceBPtr, K * N * sizeof(float));
  hipMemcpy(deviceBPtr, B.data(), K * N * sizeof(float),
             hipMemcpyHostToDevice);
  hipMalloc(&deviceCPtr, M * N * sizeof(float));
  hipMemcpy(deviceCPtr, C.data(), M * N * sizeof(float),
             hipMemcpyHostToDevice);
  hipEvent_t startEvent, stopEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);
  hipEventRecord(startEvent);
  gemmNaive(deviceAPrt, deviceBPtr, deviceCPtr, alpha, beta, M, N, K);
  hipEventRecord(stopEvent);
  hipEventSynchronize(stopEvent);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, startEvent, stopEvent);
  printf("GPU use: %.3f(ms)\n", milliseconds);
  hipEventDestroy(stopEvent);
  hipEventDestroy(startEvent);
  Matrix<float, Dynamic, Dynamic, RowMajor> hostResult{M, N},
      deviceResult{M, N};
  clock_t begin, end;
  begin = clock();
  hostResult = alpha * (A * B) + beta * C;
  end = clock();
  printf("CPU use: %.3f(ms)\n", double(end - begin) / CLOCKS_PER_SEC * 1e3);
  hipMemcpy(deviceResult.data(), deviceCPtr, M * N * sizeof(float),
             hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  Eigen::Array<float, Eigen::Dynamic, Eigen::Dynamic> diffArray =
      (hostResult - deviceResult).array().abs();
  printf("Max Error: %f\n", diffArray.maxCoeff());

  double GFLOPS = 2 * 1e-9 * M * N * K / (milliseconds * 1e-3);
  printf("GPU Throughput: %.3f GFLOPS\n", GFLOPS);
}
