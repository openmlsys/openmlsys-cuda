#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#include <Eigen/Core>
#include <argparse/argparse.hpp>
#include <ctime>
#include <iostream>
#include <utility>

void gemmFinal(const float *deviceAPtr, const float *deviceBPtr,
               float *deviceCPtr, float alpha, float beta, unsigned M,
               unsigned N, unsigned K);

void gemmUse128(const float *deviceAPtr, const float *deviceBPtr,
                float *deviceCPtr, float alpha, float beta, unsigned M,
                unsigned N, unsigned K);

void gemmUseTile(const float *deviceAPtr, const float *deviceBPtr,
                 float *deviceCPtr, float alpha, float beta, unsigned M,
                 unsigned N, unsigned K);

void gemmNaive(const float *deviceAPtr, const float *deviceBPtr,
               float *deviceCPtr, float alpha, float beta, unsigned M,
               unsigned N, unsigned K);

void gemmHideSmemLatency(const float *deviceAPtr, const float *deviceBPtr,
                         float *deviceCPtr, float alpha, float beta, unsigned M,
                         unsigned N, unsigned K);

void gemmTransposeSmem(const float *deviceAPtr, const float *deviceBPtr,
                       float *deviceCPtr, float alpha, float beta, unsigned M,
                       unsigned N, unsigned K);

void gemmUseSmem(const float *deviceAPtr, const float *deviceBPtr,
                 float *deviceCPtr, float alpha, float beta, unsigned M,
                 unsigned N, unsigned K);

class GemmTester {
  class cuTimer {
    hipEvent_t startEvent{}, stopEvent{};

  public:
    cuTimer() {
      hipEventCreate(&startEvent);
      hipEventCreate(&stopEvent);
    }
    ~cuTimer() {
      hipEventDestroy(stopEvent);
      hipEventDestroy(startEvent);
    }

    void start() { hipEventRecord(startEvent); }

    float end() {
      hipEventRecord(stopEvent);
      auto error = hipEventSynchronize(stopEvent);
      if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
      }
      float milliseconds = 0;
      hipEventElapsedTime(&milliseconds, startEvent, stopEvent);

      return milliseconds;
    }
  };

  cuTimer timer{};
  Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> hostC;
  Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor>
      deviceCCopied;
  const float *deviceAPtr, *deviceBPtr;
  float *deviceCPtr;
  const float *deviceCInitPtr;
  float alpha, beta;
  unsigned M, N, K;
  int iteration;

  void tearUp() {
    hipMemcpy(deviceCPtr, deviceCInitPtr, M * N * sizeof(float),
               hipMemcpyDeviceToDevice);
  }

  void checkValue() const {
    Eigen::Array<float, Eigen::Dynamic, Eigen::Dynamic> diffArray =
        (hostC - deviceCCopied).array().abs();

    printf("Max Error: %f\n", diffArray.maxCoeff());
  }

  template <typename Function> void profile(Function &&gemmFunction) {
    double elapsedTime = 0;
    for (int i = 0; i < iteration; ++i) {
      tearUp();
      timer.start();
      gemmFunction(deviceAPtr, deviceBPtr, deviceCPtr, alpha, beta, M, N, K);
      elapsedTime += timer.end();
    }
    elapsedTime /= iteration;
    double GFLOPS = 2 * 1e-9 * M * N * K / (elapsedTime * 1e-3);
    printf("Average Time: %.3f ms, Throughput: %.3f GFLOPS\n", elapsedTime,
           GFLOPS);
  }

public:
  explicit GemmTester(float alpha, float beta, unsigned M, unsigned N,
                      unsigned K, int iteration)
      : hostC{M, N}, deviceCCopied{M, N}, alpha(alpha), beta(beta), M(M), N(N),
        K(K), iteration{iteration} {
    Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> A{M,
                                                                            K};
    Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> B{K,
                                                                            N};
    A.setRandom();
    B.setRandom();
    hostC.setRandom();

    float *_deviceCPtr, *_deviceCInitPtr;
    hipMalloc(&_deviceCPtr, M * N * sizeof(float));
    hipMalloc(&_deviceCInitPtr, M * N * sizeof(float));
    deviceCPtr = _deviceCPtr;
    deviceCInitPtr = _deviceCInitPtr;
    hipMemcpy(_deviceCInitPtr, hostC.data(), M * N * sizeof(float),
               hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    clock_t begin, end;
    begin = clock();
    hostC = alpha * (A * B) + beta * hostC;
    end = clock();
    printf("CPU use: %.3f ms\n", double(end - begin) / CLOCKS_PER_SEC * 1e3);

    float *_deviceAPtr, *_deviceBPtr;
    hipMalloc(&_deviceAPtr, M * K * sizeof(float));
    hipMalloc(&_deviceBPtr, K * N * sizeof(float));
    hipMemcpy(_deviceAPtr, A.data(), M * K * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(_deviceBPtr, B.data(), K * N * sizeof(float),
               hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    deviceAPtr = _deviceAPtr;
    deviceBPtr = _deviceBPtr;
  }
  ~GemmTester() {
    hipFree((void *)deviceAPtr);
    hipFree((void *)deviceBPtr);
    hipFree(deviceCPtr);
  }

  template <typename Function>
  void evaluate(Function &&gemmFunction, const char *name) {
    tearUp();
    printf("-----------------------------------\n");
    printf("Evaluating %s\n", name);
    gemmFunction(deviceAPtr, deviceBPtr, deviceCPtr, alpha, beta, M, N, K);
    hipMemcpy(deviceCCopied.data(), deviceCPtr, M * N * sizeof(float),
               hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    checkValue();
    profile(std::forward<Function>(gemmFunction));
    printf("-----------------------------------\n");
  }
};

class gemmCuBlas {
  hipblasHandle_t handle{nullptr};

public:
  gemmCuBlas() { hipblasCreate(&handle); }
  ~gemmCuBlas() { hipblasDestroy(handle); }

  void operator()(const float *A, const float *B, float *C, float &alpha,
                  float &beta, int M, int N, int K) const {
    int lda = N, ldb = K, ldc = N;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, lda, A,
                ldb, &beta, C, ldc);
  }
};

int getSPcores(hipDeviceProp_t devProp) {
  int cores = 0;
  int mp = devProp.multiProcessorCount;
  switch (devProp.major) {
  case 2: // Fermi
    if (devProp.minor == 1)
      cores = mp * 48;
    else
      cores = mp * 32;
    break;
  case 3: // Kepler
    cores = mp * 192;
    break;
  case 5: // Maxwell
    cores = mp * 128;
    break;
  case 6: // Pascal
    if ((devProp.minor == 1) || (devProp.minor == 2))
      cores = mp * 128;
    else if (devProp.minor == 0)
      cores = mp * 64;
    else
      throw std::runtime_error("Unknown device type");
    break;
  case 7: // Volta and Turing
    if ((devProp.minor == 0) || (devProp.minor == 5))
      cores = mp * 64;
    else
      throw std::runtime_error("Unknown device type");
    break;
  case 8: // Ampere
    if (devProp.minor == 0)
      cores = mp * 64;
    else if (devProp.minor == 6)
      cores = mp * 128;
    else
      throw std::runtime_error("Unknown device type");
  default:
    throw std::runtime_error("Unknown device type");
  }
  return cores;
}

int main(int argc, char *argv[]) {
  argparse::ArgumentParser program("gemm");

  program.add_argument("--cpu_procs")
      .help("processor num used of CPU")
      .scan<'i', int>()
      .default_value(omp_get_num_procs());

  program.add_argument("--gpu_rank")
      .help("the used GPU rank")
      .scan<'i', int>()
      .default_value(0);

  program.add_argument("--repeat_iterations")
      .help("repeat iteration numbers and average the result")
      .scan<'i', int>()
      .default_value(10);

  program.add_argument("--alpha").scan<'g', float>();

  program.add_argument("--beta").scan<'g', float>();

  program.add_argument("-M").scan<'i', unsigned>();

  program.add_argument("-N").scan<'i', unsigned>();

  program.add_argument("-K").scan<'i', unsigned>();

  try {
    program.parse_args(argc, argv);
  } catch (const std::runtime_error &err) {
    std::cerr << err.what() << std::endl;
    std::cerr << program;
    std::exit(1);
  }

  const int cpu_procs = program.get<int>("--cpu_procs");
  const int gpu_rank = program.get<int>("--gpu_rank");
  const int repeat_iterations = program.get<int>("--repeat_iterations");
  const auto alpha = program.get<float>("--alpha");
  const auto beta = program.get<float>("--beta");
  const auto M = program.get<unsigned>("-M");
  const auto N = program.get<unsigned>("-N");
  const auto K = program.get<unsigned>("-K");
  printf("Program start with %d CPU processes on the %d-th GPU\n", cpu_procs,
         gpu_rank);
  omp_set_num_threads(cpu_procs);
  hipDeviceProp_t deviceProp{};
  hipGetDeviceProperties(&deviceProp, gpu_rank);
  hipSetDevice(gpu_rank);
  printf("GPU %s status: ", deviceProp.name);
  double boostFrequency = deviceProp.clockRate / 1e6;
  int fp32CoresNum = getSPcores(deviceProp);
  double peakPerformance = boostFrequency * fp32CoresNum * 2;
  printf("clock rate %.3f GHz, FP32 cores num %d, FP32 peak throughput %.3f "
         "GFLOPS\n",
         boostFrequency, fp32CoresNum, peakPerformance);
  printf("A: %d x %d, B: %d x %d, C: %d x %d\n", M, K, K, N, M, N);

  GemmTester tester{alpha, beta, M, N, K, repeat_iterations};
  tester.evaluate(gemmCuBlas{}, "cuBlas");
  tester.evaluate(gemmNaive, "Naive");
  tester.evaluate(gemmUse128, "Use128");
  tester.evaluate(gemmUseTile, "UseTile");
  tester.evaluate(gemmUseSmem, "UseSmem");
  tester.evaluate(gemmTransposeSmem, "TransposeSmem");
  tester.evaluate(gemmHideSmemLatency, "HideSmemLatency");
  tester.evaluate(gemmFinal, "Final");

  return 0;
}
