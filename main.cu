#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <gflags/gflags.h>
#include <omp.h>

#include <Eigen/Core>
#include <ctime>
#include <iostream>
#include <utility>

#define declGemmFn(name)                                            \
  void name(const float *deviceAPtr, const float *deviceBPtr,       \
            float *deviceCPtr, float alpha, float beta, unsigned M, \
            unsigned N, unsigned K)

declGemmFn(gemmFinal);
declGemmFn(gemmUse128);
declGemmFn(gemmUseTile);
declGemmFn(gemmNaive);
declGemmFn(gemmHideSmemLatency);
declGemmFn(gemmTransposeSmem);
declGemmFn(gemmUseSmem);

class GemmTester {
  class cuTimer {
    hipEvent_t startEvent{}, stopEvent{};

   public:
    cuTimer() {
      hipEventCreate(&startEvent);
      hipEventCreate(&stopEvent);
    }
    ~cuTimer() {
      hipEventDestroy(stopEvent);
      hipEventDestroy(startEvent);
    }

    void start() { hipEventRecord(startEvent); }

    float end() {
      hipEventRecord(stopEvent);
      auto error = hipEventSynchronize(stopEvent);
      if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
      }
      float milliseconds = 0;
      hipEventElapsedTime(&milliseconds, startEvent, stopEvent);

      return milliseconds;
    }
  };

  cuTimer timer{};
  Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> hostC;
  Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor>
      deviceCCopied;
  const float *deviceAPtr, *deviceBPtr;
  float *deviceCPtr;
  const float *deviceCInitPtr;
  float alpha, beta;
  unsigned M, N, K;
  int iteration;

  void tearUp() {
    hipMemcpy(deviceCPtr, deviceCInitPtr, M * N * sizeof(float),
               hipMemcpyDeviceToDevice);
  }

  void checkValue() const {
    Eigen::Array<float, Eigen::Dynamic, Eigen::Dynamic> diffArray =
        (hostC - deviceCCopied).array().abs();

    printf("Max Error: %f\n", diffArray.maxCoeff());
  }

  template <typename Function>
  void profile(Function &&gemmFunction) {
    double elapsedTime = 0;
    for (int i = 0; i < iteration; ++i) {
      tearUp();
      timer.start();
      gemmFunction(deviceAPtr, deviceBPtr, deviceCPtr, alpha, beta, M, N, K);
      elapsedTime += timer.end();
    }
    elapsedTime /= iteration;
    double GFLOPS = 2 * 1e-9 * M * N * K / (elapsedTime * 1e-3);
    printf("Average Time: %.3f ms, Throughput: %.3f GFLOPS\n", elapsedTime,
           GFLOPS);
  }

 public:
  explicit GemmTester(float alpha, float beta, unsigned M, unsigned N,
                      unsigned K, int iteration)
      : hostC{M, N},
        deviceCCopied{M, N},
        alpha(alpha),
        beta(beta),
        M(M),
        N(N),
        K(K),
        iteration{iteration} {
    Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> A{M,
                                                                            K};
    Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> B{K,
                                                                            N};
    A.setRandom();
    B.setRandom();
    hostC.setRandom();

    float *_deviceCPtr, *_deviceCInitPtr;
    hipMalloc(&_deviceCPtr, M * N * sizeof(float));
    hipMalloc(&_deviceCInitPtr, M * N * sizeof(float));
    deviceCPtr = _deviceCPtr;
    deviceCInitPtr = _deviceCInitPtr;
    hipMemcpy(_deviceCInitPtr, hostC.data(), M * N * sizeof(float),
               hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    clock_t begin, end;
    begin = clock();
    hostC = alpha * (A * B) + beta * hostC;
    end = clock();
    printf("CPU use: %.3f ms\n", double(end - begin) / CLOCKS_PER_SEC * 1e3);

    float *_deviceAPtr, *_deviceBPtr;
    hipMalloc(&_deviceAPtr, M * K * sizeof(float));
    hipMalloc(&_deviceBPtr, K * N * sizeof(float));
    hipMemcpy(_deviceAPtr, A.data(), M * K * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(_deviceBPtr, B.data(), K * N * sizeof(float),
               hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    deviceAPtr = _deviceAPtr;
    deviceBPtr = _deviceBPtr;
  }
  ~GemmTester() {
    hipFree((void *)deviceAPtr);
    hipFree((void *)deviceBPtr);
    hipFree(deviceCPtr);
  }

  template <typename Function>
  void evaluate(Function &&gemmFunction, const char *name) {
    tearUp();
    printf("-----------------------------------\n");
    printf("Evaluating %s\n", name);
    gemmFunction(deviceAPtr, deviceBPtr, deviceCPtr, alpha, beta, M, N, K);
    hipMemcpy(deviceCCopied.data(), deviceCPtr, M * N * sizeof(float),
               hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    checkValue();
    profile(std::forward<Function>(gemmFunction));
    printf("-----------------------------------\n");
  }
};

class gemmCuBlas {
  hipblasHandle_t handle{nullptr};

 public:
  gemmCuBlas() { hipblasCreate(&handle); }
  ~gemmCuBlas() { hipblasDestroy(handle); }

  void operator()(const float *A, const float *B, float *C, float &alpha,
                  float &beta, int M, int N, int K) const {
    int lda = N, ldb = K, ldc = N;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, lda, A,
                ldb, &beta, C, ldc);
  }
};

int getSPcores(hipDeviceProp_t devProp) {
  int cores = 0;
  int mp = devProp.multiProcessorCount;
  switch (devProp.major) {
    case 2:  // Fermi
      if (devProp.minor == 1)
        cores = mp * 48;
      else
        cores = mp * 32;
      break;
    case 3:  // Kepler
      cores = mp * 192;
      break;
    case 5:  // Maxwell
      cores = mp * 128;
      break;
    case 6:  // Pascal
      if ((devProp.minor == 1) || (devProp.minor == 2))
        cores = mp * 128;
      else if (devProp.minor == 0)
        cores = mp * 64;
      else
        throw std::runtime_error("Unknown device type");
      break;
    case 7:  // Volta and Turing
      if ((devProp.minor == 0) || (devProp.minor == 5))
        cores = mp * 64;
      else
        throw std::runtime_error("Unknown device type");
      break;
    case 8:  // Ampere
      if (devProp.minor == 0)
        cores = mp * 64;
      else if (devProp.minor == 6)
        cores = mp * 128;
      else
        throw std::runtime_error("Unknown device type");
      break;
    default:
      throw std::runtime_error("Unknown device type");
  }
  return cores;
}

DEFINE_int32(cpu_procs, omp_get_num_procs(), "processor num used of CPU");
DEFINE_int32(gpu_rank, 0, "the used GPU rank");
DEFINE_int32(repeat_iterations, 10,
             "repeat iteration numbers and average the result");
DEFINE_double(alpha, 1., "alpha");
DEFINE_double(beta, 1., "beta");
DEFINE_uint32(M, {}, "M");
DEFINE_uint32(N, {}, "N");
DEFINE_uint32(K, {}, "K");

int main(int argc, char *argv[]) {
  GFLAGS_NAMESPACE::ParseCommandLineFlags(&argc, &argv, true);

  printf("Program start with %d CPU processes on the %d-th GPU\n",
         FLAGS_cpu_procs, FLAGS_gpu_rank);
  omp_set_num_threads(FLAGS_cpu_procs);
  hipDeviceProp_t deviceProp{};
  hipGetDeviceProperties(&deviceProp, FLAGS_gpu_rank);
  hipSetDevice(FLAGS_gpu_rank);
  printf("GPU %s status: ", deviceProp.name);
  double boostFrequency = deviceProp.clockRate / 1e6;
  int fp32CoresNum = getSPcores(deviceProp);
  double peakPerformance = boostFrequency * fp32CoresNum * 2;
  printf(
      "clock rate %.3f GHz, FP32 cores num %d, FP32 peak throughput %.3f "
      "GFLOPS\n",
      boostFrequency, fp32CoresNum, peakPerformance);
  printf("A: %d x %d, B: %d x %d, C: %d x %d\n", FLAGS_M, FLAGS_K, FLAGS_K,
         FLAGS_N, FLAGS_M, FLAGS_N);

  GemmTester tester{
      (float)FLAGS_alpha,     (float)FLAGS_beta, FLAGS_M, FLAGS_N, FLAGS_K,
      FLAGS_repeat_iterations};
  tester.evaluate(gemmCuBlas{}, "cuBlas");
  tester.evaluate(gemmNaive, "Naive");
  tester.evaluate(gemmUse128, "Use128");
  tester.evaluate(gemmUseTile, "UseTile");
  tester.evaluate(gemmUseSmem, "UseSmem");
  tester.evaluate(gemmTransposeSmem, "TransposeSmem");
  tester.evaluate(gemmHideSmemLatency, "HideSmemLatency");
  tester.evaluate(gemmFinal, "Final");

  GFLAGS_NAMESPACE::ShutDownCommandLineFlags();
  return 0;
}
