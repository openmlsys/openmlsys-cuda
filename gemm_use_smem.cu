#include "hip/hip_runtime.h"
#include "util.cuh"

namespace {
template <typename LayoutTile, typename LayoutBlock, typename LayoutThread>
__global__ void gemmKernel(const float *__restrict__ A,
                           const float *__restrict__ B, float *__restrict__ C,
                           float alpha, float beta, unsigned M, unsigned N,
                           unsigned K) {
  constexpr unsigned ratio = sizeof(openmlsys::float4) / sizeof(float);
  using LayoutTileT =
      openmlsys::Layout<LayoutTile::m / ratio, LayoutTile::n / ratio,
                                LayoutTile::k / ratio>;
  using LayoutThreadT =
      openmlsys::Layout<LayoutThread::m / ratio, LayoutThread::n / ratio>;
  constexpr unsigned blockSize = LayoutBlock::m * LayoutBlock::n;
  constexpr openmlsys::float4 float4Zero{0.f, 0.f, 0.f, 0.f};

  __shared__ openmlsys::float4 tileA[LayoutTile::m][LayoutTileT::k];
  __shared__ openmlsys::float4 tileB[LayoutTile::k][LayoutTileT::n];

  const unsigned nInTileC = threadIdx.x % LayoutBlock::m;
  const unsigned mInTileC = threadIdx.x / LayoutBlock::m;

  const unsigned kInTileA = threadIdx.x % LayoutTileT::k;
  const unsigned mInTileA = threadIdx.x / LayoutTileT::k;

  const unsigned nInTileB = threadIdx.x % LayoutTileT::n;
  const unsigned kinTileB = threadIdx.x / LayoutTileT::n;

  openmlsys::Tensor2D<const openmlsys::float4> pA{A, M, K / ratio};
  pA.addOffset(LayoutTile::m * blockIdx.y + mInTileA, kInTileA);
  openmlsys::Tensor2D<const openmlsys::float4> pB{B, K, N / ratio};
  pB.addOffset(kinTileB,
               LayoutTileT::n * blockIdx.x + nInTileB * LayoutThreadT::n);
  openmlsys::Tensor2D<openmlsys::float4> pC{C, M, N / ratio};
  pC.addOffset(LayoutTile::m * blockIdx.y + mInTileC * LayoutThread::m,
               LayoutTileT::n * blockIdx.x + nInTileC * LayoutThreadT::n);

  constexpr unsigned tileSizeA = LayoutTile::m * LayoutTile::k;
  constexpr unsigned tileSizeB = LayoutTile::n * LayoutTile::k;
  constexpr unsigned tileIterationsA = tileSizeA / blockSize / ratio;
  constexpr unsigned tileGlobalIntervalA = blockSize / LayoutTileT::k;
  constexpr unsigned tileComputeIterationsA = LayoutTileT::m / LayoutBlock::m;
  constexpr unsigned tileSharedIntervalA = LayoutTile::m / tileComputeIterationsA;
  constexpr unsigned tileIterationsB = tileSizeB / blockSize / ratio;
  constexpr unsigned tileGlobalIntervalB = blockSize / LayoutTileT::n;
  constexpr unsigned tileComputeIterationsB = LayoutTileT::n / LayoutBlock::n;
  constexpr unsigned tileSharedIntervalBT = LayoutTileT::n / tileComputeIterationsB;

  openmlsys::float4 bufferA[tileIterationsA];
  openmlsys::float4 bufferB[tileIterationsB];
  bool validLoadTileA[tileIterationsA];
  bool validLoadTileB[tileIterationsB];

#pragma unroll
  for (unsigned i = 0; i < tileIterationsA; ++i) {
    validLoadTileA[i] = pA.validRowOffset(i * tileGlobalIntervalA);
  }

#pragma unroll
  for (unsigned i = 0; i < tileIterationsB; ++i) {
    validLoadTileB[i] = pB.validColOffset(0);
  }

  openmlsys::float4 c[tileComputeIterationsA * LayoutThread::m]
             [tileComputeIterationsB * LayoutThreadT::n];
  memset(c, 0, sizeof(c));

  openmlsys::float4 fragmentA[tileComputeIterationsA * LayoutThreadT::m];
  openmlsys::float4 fragmentB[tileComputeIterationsB * LayoutThreadT::n];

  for (unsigned i = 0; i < K; i += LayoutTile::k) {
#pragma unroll
    for (unsigned j = 0; j < tileIterationsA; ++j) {
      validLoadTileA[j] = validLoadTileA[j] && pA.validColOffset(0);
      bufferA[j] =
          validLoadTileA[j] ? pA(j * tileGlobalIntervalA, 0) : float4Zero;
    }

#pragma unroll
    for (unsigned j = 0; j < tileIterationsB; ++j) {
      validLoadTileB[j] =
          validLoadTileB[j] && pB.validRowOffset(j * tileGlobalIntervalB);
      bufferB[j] =
          validLoadTileB[j] ? pB(j * tileGlobalIntervalB, 0) : float4Zero;
    }

    __syncthreads();
#pragma unroll
    for (unsigned a = 0; a < tileIterationsA; ++a) {
      tileA[mInTileA + a * tileGlobalIntervalA][kInTileA] = bufferA[a];
    }

#pragma unroll
    for (unsigned a = 0; a < tileIterationsB; ++a) {
      tileB[kinTileB + a * tileGlobalIntervalB][nInTileB] = bufferB[a];
    }
    __syncthreads();

#pragma unroll
    for (unsigned j = 0; j < LayoutTile::k; j++) {
#pragma unroll
      for (unsigned a = 0; a < tileComputeIterationsA; ++a) {
#pragma unroll
        for (unsigned b = 0; b < LayoutThread::m; ++b) {
          fragmentA[a][b] =
              tileA[a * tileSharedIntervalA + mInTileC * LayoutThread::m + b]
                   [j / ratio][j % ratio];
        }
      }
#pragma unroll
      for (unsigned a = 0; a < tileComputeIterationsB; ++a) {
        fragmentB[a] = tileB[j][a * tileSharedIntervalBT + nInTileC];
      }
#pragma unroll
      for (unsigned d = 0; d < tileComputeIterationsA * LayoutThread::m; ++d) {
#pragma unroll
        for (unsigned e = 0; e < tileComputeIterationsB * LayoutThreadT::n; ++e) {
          c[d][e] =
              c[d][e] + fragmentB[e] *
                            fragmentA[d / LayoutThread::m][d % LayoutThread::m];
        }
      }
    }
    pA.addOffset(0, LayoutTileT::k);
    pB.addOffset(LayoutTile::k, 0);
  }

#pragma unroll
  for (auto &a : c) {
#pragma unroll
    for (auto &b : a) {
      b = b * alpha;
    }
  }

#pragma unroll
  for (unsigned i = 0; i < tileComputeIterationsA; ++i) {
#pragma unroll
    for (unsigned a = 0; a < LayoutThread::m; a++) {
      const bool mValid = pC.validRowOffset(a);
#pragma unroll
      for (unsigned b = 0; b < tileComputeIterationsB; b++) {
        const bool nValid = pC.validColOffset(b * tileSharedIntervalBT);
        if (mValid && nValid) {
          openmlsys::float4 result{c[a + i * LayoutThread::m][b]};
          if (beta != 0) {
            result = result + pC(a, b * tileSharedIntervalBT) * beta;
          }
          pC(a, b * tileSharedIntervalBT) = result;
        }
      }
    }
    pC.addOffset(tileSharedIntervalA, 0);
  }
}
}  // namespace

void gemmUseSmem(const float *deviceAPtr, const float *deviceBPtr,
                 float *deviceCPtr, float alpha, float beta, unsigned M,
                 unsigned N, unsigned K) {
  using LayoutTile = openmlsys::Layout<128, 128, 16>;
  using LayoutBlock = openmlsys::Layout<16, 16>;
  using LayoutThread = openmlsys::Layout<4, 4>;

  dim3 block(LayoutBlock::m * LayoutBlock::n);
  dim3 grid((M - 1) / LayoutTile::m + 1, (N - 1) / LayoutTile::n + 1);

  gemmKernel<LayoutTile, LayoutBlock, LayoutThread><<<grid, block>>>(
      deviceAPtr, deviceBPtr, deviceCPtr, alpha, beta, M, N, K);
}
