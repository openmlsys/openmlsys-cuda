#include "hip/hip_runtime.h"
#include "util.cuh"

namespace {
__global__ void gemmKernel(const float *__restrict__ A,
                           const float *__restrict__ B, float *__restrict__ C,
                           float alpha, float beta, unsigned M, unsigned N,
                           unsigned K) {
  constexpr unsigned ratio = sizeof(openmlsys::float4) / sizeof(float);
  unsigned int m = (threadIdx.x + blockDim.x * blockIdx.x) * ratio;
  unsigned int n = (threadIdx.y + blockDim.y * blockIdx.y) * ratio;
  openmlsys::Tensor2D<const float> pA{A, M, K};
  pA.addOffset(m, 0);
  openmlsys::Tensor2D<const openmlsys::float4> pB{B, K, N / ratio};
  pB.addOffset(0, n / ratio);
  openmlsys::Tensor2D<openmlsys::float4> pC{C, M, N / ratio};
  pC.addOffset(m, n / ratio);
  if (!pC.validOffset(0, 0)) return;

  openmlsys::float4 c[4];
  memset(c, 0, sizeof(c));
  for (unsigned k = 0; k < K; ++k) {
    openmlsys::float4 fragmentA{};
#pragma unroll
    for (unsigned i = 0; i < ratio; ++i) {
      fragmentA[i] = pA(i, k);
    }
    openmlsys::float4 fragmentB = pB(k, 0);

#pragma unroll
    for (unsigned i = 0; i < ratio; ++i) {
      c[i] = c[i] + fragmentB * fragmentA[i];
    }
  }

#pragma unroll
  for (auto &a : c) {
    a = a * alpha;
  }

#pragma unroll
  for (unsigned i = 0; i < ratio; ++i) {
    openmlsys::float4 result = c[i];
    if (beta != 0) {
      result = c[i] + pC(i, 0) * beta;
    }
    pC(i, 0) = result;
  }
}
}  // namespace

void gemmUse128(const float *deviceAPtr, const float *deviceBPtr,
                float *deviceCPtr, float alpha, float beta, unsigned M,
                unsigned N, unsigned K) {
  dim3 block(16, 16);
  dim3 grid((M / 4 - 1) / block.x + 1, (N / 4 - 1) / block.y + 1);

  gemmKernel<<<grid, block>>>(deviceAPtr, deviceBPtr, deviceCPtr, alpha, beta,
                              M, N, K);
}
